/* Copyright 2023 CMU, Facebook, LANL, MIT, NVIDIA, and Stanford (alphabetical)
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#if defined(FF_USE_CUDA) || defined(FF_USE_HIP_CUDA)
#include "hip/hip_complex.h"
#endif
#include "flashinfer/attention_impl.cuh"

// This is for instantiating the template attention kernels
namespace flashinfer {

// group_size[] = {1, 4, 8};
// head_dim[] = {64, 128, 256};

/********** prefill instantiations for half precision **********/

template hipError_t SinglePrefillWithKVCacheDispatched<
  1, 64, QKVLayout::kNHD, PosEncodingMode::kNone,
  false, MaskMode::kCausal, half, half>(
    half* q, half* k, half* v, float* custom_mask, half* o,
    float* tmp, float* lse, uint32_t num_kv_heads, uint32_t qo_len, uint32_t kv_len,
    float sm_scale, float rope_scale,
    float rope_theta, hipStream_t stream);

template hipError_t SinglePrefillWithKVCacheDispatched<
  1, 128, QKVLayout::kNHD, PosEncodingMode::kNone,
  false, MaskMode::kCausal, half, half>(
    half* q, half* k, half* v, float* custom_mask, half* o,
    float* tmp, float* lse, uint32_t num_kv_heads, uint32_t qo_len, uint32_t kv_len,
    float sm_scale, float rope_scale,
    float rope_theta, hipStream_t stream);

template hipError_t SinglePrefillWithKVCacheDispatched<
  1, 256, QKVLayout::kNHD, PosEncodingMode::kNone,
  false, MaskMode::kCausal, half, half>(
    half* q, half* k, half* v, float* custom_mask, half* o,
    float* tmp, float* lse, uint32_t num_kv_heads, uint32_t qo_len, uint32_t kv_len,
    float sm_scale, float rope_scale,
    float rope_theta, hipStream_t stream);

template hipError_t SinglePrefillWithKVCacheDispatched<
  4, 64, QKVLayout::kNHD, PosEncodingMode::kNone,
  false, MaskMode::kCausal, half, half>(
    half* q, half* k, half* v, float* custom_mask, half* o,
    float* tmp, float* lse, uint32_t num_kv_heads, uint32_t qo_len, uint32_t kv_len,
    float sm_scale, float rope_scale,
    float rope_theta, hipStream_t stream);

template hipError_t SinglePrefillWithKVCacheDispatched<
  4, 128, QKVLayout::kNHD, PosEncodingMode::kNone,
  false, MaskMode::kCausal, half, half>(
    half* q, half* k, half* v, float* custom_mask, half* o,
    float* tmp, float* lse, uint32_t num_kv_heads, uint32_t qo_len, uint32_t kv_len,
    float sm_scale, float rope_scale,
    float rope_theta, hipStream_t stream);

template hipError_t SinglePrefillWithKVCacheDispatched<
  4, 256, QKVLayout::kNHD, PosEncodingMode::kNone,
  false, MaskMode::kCausal, half, half>(
    half* q, half* k, half* v, float* custom_mask, half* o,
    float* tmp, float* lse, uint32_t num_kv_heads, uint32_t qo_len, uint32_t kv_len,
    float sm_scale, float rope_scale,
    float rope_theta, hipStream_t stream);

template hipError_t SinglePrefillWithKVCacheDispatched<
  8, 64, QKVLayout::kNHD, PosEncodingMode::kNone,
  false, MaskMode::kCausal, half, half>(
    half* q, half* k, half* v, float* custom_mask, half* o,
    float* tmp, float* lse, uint32_t num_kv_heads, uint32_t qo_len, uint32_t kv_len,
    float sm_scale, float rope_scale,
    float rope_theta, hipStream_t stream);

template hipError_t SinglePrefillWithKVCacheDispatched<
  8, 128, QKVLayout::kNHD, PosEncodingMode::kNone,
  false, MaskMode::kCausal, half, half>(
    half* q, half* k, half* v, float* custom_mask, half* o,
    float* tmp, float* lse, uint32_t num_kv_heads, uint32_t qo_len, uint32_t kv_len,
    float sm_scale, float rope_scale,
    float rope_theta, hipStream_t stream);

template hipError_t SinglePrefillWithKVCacheDispatched<
  8, 256, QKVLayout::kNHD, PosEncodingMode::kNone,
  false, MaskMode::kCausal, half, half>(
    half* q, half* k, half* v, float* custom_mask, half* o,
    float* tmp, float* lse, uint32_t num_kv_heads, uint32_t qo_len, uint32_t kv_len,
    float sm_scale, float rope_scale,
    float rope_theta, hipStream_t stream);


/********** append instantiations for half precision **********/

template hipError_t SinglePrefillWithKVCacheDispatched<
  1, 64, QKVLayout::kNHD, PosEncodingMode::kNone,
  false, MaskMode::kCustom, half, half>(
    half* q, half* k, half* v, float* custom_mask, half* o,
    float* tmp, float* lse, uint32_t num_kv_heads, uint32_t qo_len, uint32_t kv_len,
    float sm_scale, float rope_scale,
    float rope_theta, hipStream_t stream);

template hipError_t SinglePrefillWithKVCacheDispatched<
  1, 128, QKVLayout::kNHD, PosEncodingMode::kNone,
  false, MaskMode::kCustom, half, half>(
    half* q, half* k, half* v, float* custom_mask, half* o,
    float* tmp, float* lse, uint32_t num_kv_heads, uint32_t qo_len, uint32_t kv_len,
    float sm_scale, float rope_scale,
    float rope_theta, hipStream_t stream);

template hipError_t SinglePrefillWithKVCacheDispatched<
  1, 256, QKVLayout::kNHD, PosEncodingMode::kNone,
  false, MaskMode::kCustom, half, half>(
    half* q, half* k, half* v, float* custom_mask, half* o,
    float* tmp, float* lse, uint32_t num_kv_heads, uint32_t qo_len, uint32_t kv_len,
    float sm_scale, float rope_scale,
    float rope_theta, hipStream_t stream);

template hipError_t SinglePrefillWithKVCacheDispatched<
  4, 64, QKVLayout::kNHD, PosEncodingMode::kNone,
  false, MaskMode::kCustom, half, half>(
    half* q, half* k, half* v, float* custom_mask, half* o,
    float* tmp, float* lse, uint32_t num_kv_heads, uint32_t qo_len, uint32_t kv_len,
    float sm_scale, float rope_scale,
    float rope_theta, hipStream_t stream);

template hipError_t SinglePrefillWithKVCacheDispatched<
  4, 128, QKVLayout::kNHD, PosEncodingMode::kNone,
  false, MaskMode::kCustom, half, half>(
    half* q, half* k, half* v, float* custom_mask, half* o,
    float* tmp, float* lse, uint32_t num_kv_heads, uint32_t qo_len, uint32_t kv_len,
    float sm_scale, float rope_scale,
    float rope_theta, hipStream_t stream);

template hipError_t SinglePrefillWithKVCacheDispatched<
  4, 256, QKVLayout::kNHD, PosEncodingMode::kNone,
  false, MaskMode::kCustom, half, half>(
    half* q, half* k, half* v, float* custom_mask, half* o,
    float* tmp, float* lse, uint32_t num_kv_heads, uint32_t qo_len, uint32_t kv_len,
    float sm_scale, float rope_scale,
    float rope_theta, hipStream_t stream);

template hipError_t SinglePrefillWithKVCacheDispatched<
  8, 64, QKVLayout::kNHD, PosEncodingMode::kNone,
  false, MaskMode::kCustom, half, half>(
    half* q, half* k, half* v, float* custom_mask, half* o,
    float* tmp, float* lse, uint32_t num_kv_heads, uint32_t qo_len, uint32_t kv_len,
    float sm_scale, float rope_scale,
    float rope_theta, hipStream_t stream);

template hipError_t SinglePrefillWithKVCacheDispatched<
  8, 128, QKVLayout::kNHD, PosEncodingMode::kNone,
  false, MaskMode::kCustom, half, half>(
    half* q, half* k, half* v, float* custom_mask, half* o,
    float* tmp, float* lse, uint32_t num_kv_heads, uint32_t qo_len, uint32_t kv_len,
    float sm_scale, float rope_scale,
    float rope_theta, hipStream_t stream);

template hipError_t SinglePrefillWithKVCacheDispatched<
  8, 256, QKVLayout::kNHD, PosEncodingMode::kNone,
  false, MaskMode::kCustom, half, half>(
    half* q, half* k, half* v, float* custom_mask, half* o,
    float* tmp, float* lse, uint32_t num_kv_heads, uint32_t qo_len, uint32_t kv_len,
    float sm_scale, float rope_scale,
    float rope_theta, hipStream_t stream);


constexpr uint32_t kPagesize = 64;
// num_frags_x[] = {1, 2};
// group_size[] = {1, 4, 8};
// head_dim[] = {64, 128, 256};

/********** batch append instantiations for half precision **********/

template hipError_t BatchPrefillWithPagedKVCacheDispatched<
  PageStorage::kIndices, QKVLayout::kNHD, 1, kPagesize,
  1, 64, PosEncodingMode::kNone, false, MaskMode::kCustom,
  half, half, int32_t>(
    half* q, int32_t* request_indices, int32_t* tile_indices, int32_t* qo_indptr, int32_t* q_offset,
    paged_kv_t<PageStorage::kIndices, QKVLayout::kNHD, half, int32_t> paged_kv, float* custom_mask,
    int32_t* qk_indptr, half* o, float* tmp, float* lse, uint32_t num_qo_tiles, float sm_scale,
    float rope_scale, float rope_theta, hipStream_t stream);

template hipError_t BatchPrefillWithPagedKVCacheDispatched<
  PageStorage::kIndices, QKVLayout::kNHD, 1, kPagesize,
  1, 128, PosEncodingMode::kNone, false, MaskMode::kCustom,
  half, half, int32_t>(
    half* q, int32_t* request_indices, int32_t* tile_indices, int32_t* qo_indptr, int32_t* q_offset,
    paged_kv_t<PageStorage::kIndices, QKVLayout::kNHD, half, int32_t> paged_kv, float* custom_mask,
    int32_t* qk_indptr, half* o, float* tmp, float* lse, uint32_t num_qo_tiles, float sm_scale,
    float rope_scale, float rope_theta, hipStream_t stream);

template hipError_t BatchPrefillWithPagedKVCacheDispatched<
  PageStorage::kIndices, QKVLayout::kNHD, 1, kPagesize,
  1, 256, PosEncodingMode::kNone, false, MaskMode::kCustom,
  half, half, int32_t>(
    half* q, int32_t* request_indices, int32_t* tile_indices, int32_t* qo_indptr, int32_t* q_offset,
    paged_kv_t<PageStorage::kIndices, QKVLayout::kNHD, half, int32_t> paged_kv, float* custom_mask,
    int32_t* qk_indptr, half* o, float* tmp, float* lse, uint32_t num_qo_tiles, float sm_scale,
    float rope_scale, float rope_theta, hipStream_t stream);

template hipError_t BatchPrefillWithPagedKVCacheDispatched<
  PageStorage::kIndices, QKVLayout::kNHD, 1, kPagesize,
  4, 64, PosEncodingMode::kNone, false, MaskMode::kCustom,
  half, half, int32_t>(
    half* q, int32_t* request_indices, int32_t* tile_indices, int32_t* qo_indptr, int32_t* q_offset,
    paged_kv_t<PageStorage::kIndices, QKVLayout::kNHD, half, int32_t> paged_kv, float* custom_mask,
    int32_t* qk_indptr, half* o, float* tmp, float* lse, uint32_t num_qo_tiles, float sm_scale,
    float rope_scale, float rope_theta, hipStream_t stream);

template hipError_t BatchPrefillWithPagedKVCacheDispatched<
  PageStorage::kIndices, QKVLayout::kNHD, 1, kPagesize,
  4, 128, PosEncodingMode::kNone, false, MaskMode::kCustom,
  half, half, int32_t>(
    half* q, int32_t* request_indices, int32_t* tile_indices, int32_t* qo_indptr, int32_t* q_offset,
    paged_kv_t<PageStorage::kIndices, QKVLayout::kNHD, half, int32_t> paged_kv, float* custom_mask,
    int32_t* qk_indptr, half* o, float* tmp, float* lse, uint32_t num_qo_tiles, float sm_scale,
    float rope_scale, float rope_theta, hipStream_t stream);

template hipError_t BatchPrefillWithPagedKVCacheDispatched<
  PageStorage::kIndices, QKVLayout::kNHD, 1, kPagesize,
  4, 256, PosEncodingMode::kNone, false, MaskMode::kCustom,
  half, half, int32_t>(
    half* q, int32_t* request_indices, int32_t* tile_indices, int32_t* qo_indptr, int32_t* q_offset,
    paged_kv_t<PageStorage::kIndices, QKVLayout::kNHD, half, int32_t> paged_kv, float* custom_mask,
    int32_t* qk_indptr, half* o, float* tmp, float* lse, uint32_t num_qo_tiles, float sm_scale,
    float rope_scale, float rope_theta, hipStream_t stream);

template hipError_t BatchPrefillWithPagedKVCacheDispatched<
  PageStorage::kIndices, QKVLayout::kNHD, 1, kPagesize,
  8, 64, PosEncodingMode::kNone, false, MaskMode::kCustom,
  half, half, int32_t>(
    half* q, int32_t* request_indices, int32_t* tile_indices, int32_t* qo_indptr, int32_t* q_offset,
    paged_kv_t<PageStorage::kIndices, QKVLayout::kNHD, half, int32_t> paged_kv, float* custom_mask,
    int32_t* qk_indptr, half* o, float* tmp, float* lse, uint32_t num_qo_tiles, float sm_scale,
    float rope_scale, float rope_theta, hipStream_t stream);

template hipError_t BatchPrefillWithPagedKVCacheDispatched<
  PageStorage::kIndices, QKVLayout::kNHD, 1, kPagesize,
  8, 128, PosEncodingMode::kNone, false, MaskMode::kCustom,
  half, half, int32_t>(
    half* q, int32_t* request_indices, int32_t* tile_indices, int32_t* qo_indptr, int32_t* q_offset,
    paged_kv_t<PageStorage::kIndices, QKVLayout::kNHD, half, int32_t> paged_kv, float* custom_mask,
    int32_t* qk_indptr, half* o, float* tmp, float* lse, uint32_t num_qo_tiles, float sm_scale,
    float rope_scale, float rope_theta, hipStream_t stream);

template hipError_t BatchPrefillWithPagedKVCacheDispatched<
  PageStorage::kIndices, QKVLayout::kNHD, 1, kPagesize,
  8, 256, PosEncodingMode::kNone, false, MaskMode::kCustom,
  half, half, int32_t>(
    half* q, int32_t* request_indices, int32_t* tile_indices, int32_t* qo_indptr, int32_t* q_offset,
    paged_kv_t<PageStorage::kIndices, QKVLayout::kNHD, half, int32_t> paged_kv, float* custom_mask,
    int32_t* qk_indptr, half* o, float* tmp, float* lse, uint32_t num_qo_tiles, float sm_scale,
    float rope_scale, float rope_theta, hipStream_t stream);

template hipError_t BatchPrefillWithPagedKVCacheDispatched<
  PageStorage::kIndices, QKVLayout::kNHD, 2, kPagesize,
  1, 64, PosEncodingMode::kNone, false, MaskMode::kCustom,
  half, half, int32_t>(
    half* q, int32_t* request_indices, int32_t* tile_indices, int32_t* qo_indptr, int32_t* q_offset,
    paged_kv_t<PageStorage::kIndices, QKVLayout::kNHD, half, int32_t> paged_kv, float* custom_mask,
    int32_t* qk_indptr, half* o, float* tmp, float* lse, uint32_t num_qo_tiles, float sm_scale,
    float rope_scale, float rope_theta, hipStream_t stream);

template hipError_t BatchPrefillWithPagedKVCacheDispatched<
  PageStorage::kIndices, QKVLayout::kNHD, 2, kPagesize,
  1, 128, PosEncodingMode::kNone, false, MaskMode::kCustom,
  half, half, int32_t>(
    half* q, int32_t* request_indices, int32_t* tile_indices, int32_t* qo_indptr, int32_t* q_offset,
    paged_kv_t<PageStorage::kIndices, QKVLayout::kNHD, half, int32_t> paged_kv, float* custom_mask,
    int32_t* qk_indptr, half* o, float* tmp, float* lse, uint32_t num_qo_tiles, float sm_scale,
    float rope_scale, float rope_theta, hipStream_t stream);

template hipError_t BatchPrefillWithPagedKVCacheDispatched<
  PageStorage::kIndices, QKVLayout::kNHD, 2, kPagesize,
  1, 256, PosEncodingMode::kNone, false, MaskMode::kCustom,
  half, half, int32_t>(
    half* q, int32_t* request_indices, int32_t* tile_indices, int32_t* qo_indptr, int32_t* q_offset,
    paged_kv_t<PageStorage::kIndices, QKVLayout::kNHD, half, int32_t> paged_kv, float* custom_mask,
    int32_t* qk_indptr, half* o, float* tmp, float* lse, uint32_t num_qo_tiles, float sm_scale,
    float rope_scale, float rope_theta, hipStream_t stream);

template hipError_t BatchPrefillWithPagedKVCacheDispatched<
  PageStorage::kIndices, QKVLayout::kNHD, 2, kPagesize,
  4, 64, PosEncodingMode::kNone, false, MaskMode::kCustom,
  half, half, int32_t>(
    half* q, int32_t* request_indices, int32_t* tile_indices, int32_t* qo_indptr, int32_t* q_offset,
    paged_kv_t<PageStorage::kIndices, QKVLayout::kNHD, half, int32_t> paged_kv, float* custom_mask,
    int32_t* qk_indptr, half* o, float* tmp, float* lse, uint32_t num_qo_tiles, float sm_scale,
    float rope_scale, float rope_theta, hipStream_t stream);

template hipError_t BatchPrefillWithPagedKVCacheDispatched<
  PageStorage::kIndices, QKVLayout::kNHD, 2, kPagesize,
  4, 128, PosEncodingMode::kNone, false, MaskMode::kCustom,
  half, half, int32_t>(
    half* q, int32_t* request_indices, int32_t* tile_indices, int32_t* qo_indptr, int32_t* q_offset,
    paged_kv_t<PageStorage::kIndices, QKVLayout::kNHD, half, int32_t> paged_kv, float* custom_mask,
    int32_t* qk_indptr, half* o, float* tmp, float* lse, uint32_t num_qo_tiles, float sm_scale,
    float rope_scale, float rope_theta, hipStream_t stream);

template hipError_t BatchPrefillWithPagedKVCacheDispatched<
  PageStorage::kIndices, QKVLayout::kNHD, 2, kPagesize,
  4, 256, PosEncodingMode::kNone, false, MaskMode::kCustom,
  half, half, int32_t>(
    half* q, int32_t* request_indices, int32_t* tile_indices, int32_t* qo_indptr, int32_t* q_offset,
    paged_kv_t<PageStorage::kIndices, QKVLayout::kNHD, half, int32_t> paged_kv, float* custom_mask,
    int32_t* qk_indptr, half* o, float* tmp, float* lse, uint32_t num_qo_tiles, float sm_scale,
    float rope_scale, float rope_theta, hipStream_t stream);

template hipError_t BatchPrefillWithPagedKVCacheDispatched<
  PageStorage::kIndices, QKVLayout::kNHD, 2, kPagesize,
  8, 64, PosEncodingMode::kNone, false, MaskMode::kCustom,
  half, half, int32_t>(
    half* q, int32_t* request_indices, int32_t* tile_indices, int32_t* qo_indptr, int32_t* q_offset,
    paged_kv_t<PageStorage::kIndices, QKVLayout::kNHD, half, int32_t> paged_kv, float* custom_mask,
    int32_t* qk_indptr, half* o, float* tmp, float* lse, uint32_t num_qo_tiles, float sm_scale,
    float rope_scale, float rope_theta, hipStream_t stream);

template hipError_t BatchPrefillWithPagedKVCacheDispatched<
  PageStorage::kIndices, QKVLayout::kNHD, 2, kPagesize,
  8, 128, PosEncodingMode::kNone, false, MaskMode::kCustom,
  half, half, int32_t>(
    half* q, int32_t* request_indices, int32_t* tile_indices, int32_t* qo_indptr, int32_t* q_offset,
    paged_kv_t<PageStorage::kIndices, QKVLayout::kNHD, half, int32_t> paged_kv, float* custom_mask,
    int32_t* qk_indptr, half* o, float* tmp, float* lse, uint32_t num_qo_tiles, float sm_scale,
    float rope_scale, float rope_theta, hipStream_t stream);

template hipError_t BatchPrefillWithPagedKVCacheDispatched<
  PageStorage::kIndices, QKVLayout::kNHD, 2, kPagesize,
  8, 256, PosEncodingMode::kNone, false, MaskMode::kCustom,
  half, half, int32_t>(
    half* q, int32_t* request_indices, int32_t* tile_indices, int32_t* qo_indptr, int32_t* q_offset,
    paged_kv_t<PageStorage::kIndices, QKVLayout::kNHD, half, int32_t> paged_kv, float* custom_mask,
    int32_t* qk_indptr, half* o, float* tmp, float* lse, uint32_t num_qo_tiles, float sm_scale,
    float rope_scale, float rope_theta, hipStream_t stream);


/********** batch prefill instantiations for half precision **********/

template hipError_t BatchPrefillWithPagedKVCacheDispatched<
  PageStorage::kIndices, QKVLayout::kNHD, 1, kPagesize,
  1, 64, PosEncodingMode::kNone, false, MaskMode::kCausal,
  half, half, int32_t>(
    half* q, int32_t* request_indices, int32_t* tile_indices, int32_t* qo_indptr, int32_t* q_offset,
    paged_kv_t<PageStorage::kIndices, QKVLayout::kNHD, half, int32_t> paged_kv, float* custom_mask,
    int32_t* qk_indptr, half* o, float* tmp, float* lse, uint32_t num_qo_tiles, float sm_scale,
    float rope_scale, float rope_theta, hipStream_t stream);

template hipError_t BatchPrefillWithPagedKVCacheDispatched<
  PageStorage::kIndices, QKVLayout::kNHD, 1, kPagesize,
  1, 128, PosEncodingMode::kNone, false, MaskMode::kCausal,
  half, half, int32_t>(
    half* q, int32_t* request_indices, int32_t* tile_indices, int32_t* qo_indptr, int32_t* q_offset,
    paged_kv_t<PageStorage::kIndices, QKVLayout::kNHD, half, int32_t> paged_kv, float* custom_mask,
    int32_t* qk_indptr, half* o, float* tmp, float* lse, uint32_t num_qo_tiles, float sm_scale,
    float rope_scale, float rope_theta, hipStream_t stream);

template hipError_t BatchPrefillWithPagedKVCacheDispatched<
  PageStorage::kIndices, QKVLayout::kNHD, 1, kPagesize,
  1, 256, PosEncodingMode::kNone, false, MaskMode::kCausal,
  half, half, int32_t>(
    half* q, int32_t* request_indices, int32_t* tile_indices, int32_t* qo_indptr, int32_t* q_offset,
    paged_kv_t<PageStorage::kIndices, QKVLayout::kNHD, half, int32_t> paged_kv, float* custom_mask,
    int32_t* qk_indptr, half* o, float* tmp, float* lse, uint32_t num_qo_tiles, float sm_scale,
    float rope_scale, float rope_theta, hipStream_t stream);

template hipError_t BatchPrefillWithPagedKVCacheDispatched<
  PageStorage::kIndices, QKVLayout::kNHD, 1, kPagesize,
  4, 64, PosEncodingMode::kNone, false, MaskMode::kCausal,
  half, half, int32_t>(
    half* q, int32_t* request_indices, int32_t* tile_indices, int32_t* qo_indptr, int32_t* q_offset,
    paged_kv_t<PageStorage::kIndices, QKVLayout::kNHD, half, int32_t> paged_kv, float* custom_mask,
    int32_t* qk_indptr, half* o, float* tmp, float* lse, uint32_t num_qo_tiles, float sm_scale,
    float rope_scale, float rope_theta, hipStream_t stream);

template hipError_t BatchPrefillWithPagedKVCacheDispatched<
  PageStorage::kIndices, QKVLayout::kNHD, 1, kPagesize,
  4, 128, PosEncodingMode::kNone, false, MaskMode::kCausal,
  half, half, int32_t>(
    half* q, int32_t* request_indices, int32_t* tile_indices, int32_t* qo_indptr, int32_t* q_offset,
    paged_kv_t<PageStorage::kIndices, QKVLayout::kNHD, half, int32_t> paged_kv, float* custom_mask,
    int32_t* qk_indptr, half* o, float* tmp, float* lse, uint32_t num_qo_tiles, float sm_scale,
    float rope_scale, float rope_theta, hipStream_t stream);

template hipError_t BatchPrefillWithPagedKVCacheDispatched<
  PageStorage::kIndices, QKVLayout::kNHD, 1, kPagesize,
  4, 256, PosEncodingMode::kNone, false, MaskMode::kCausal,
  half, half, int32_t>(
    half* q, int32_t* request_indices, int32_t* tile_indices, int32_t* qo_indptr, int32_t* q_offset,
    paged_kv_t<PageStorage::kIndices, QKVLayout::kNHD, half, int32_t> paged_kv, float* custom_mask,
    int32_t* qk_indptr, half* o, float* tmp, float* lse, uint32_t num_qo_tiles, float sm_scale,
    float rope_scale, float rope_theta, hipStream_t stream);

template hipError_t BatchPrefillWithPagedKVCacheDispatched<
  PageStorage::kIndices, QKVLayout::kNHD, 1, kPagesize,
  8, 64, PosEncodingMode::kNone, false, MaskMode::kCausal,
  half, half, int32_t>(
    half* q, int32_t* request_indices, int32_t* tile_indices, int32_t* qo_indptr, int32_t* q_offset,
    paged_kv_t<PageStorage::kIndices, QKVLayout::kNHD, half, int32_t> paged_kv, float* custom_mask,
    int32_t* qk_indptr, half* o, float* tmp, float* lse, uint32_t num_qo_tiles, float sm_scale,
    float rope_scale, float rope_theta, hipStream_t stream);

template hipError_t BatchPrefillWithPagedKVCacheDispatched<
  PageStorage::kIndices, QKVLayout::kNHD, 1, kPagesize,
  8, 128, PosEncodingMode::kNone, false, MaskMode::kCausal,
  half, half, int32_t>(
    half* q, int32_t* request_indices, int32_t* tile_indices, int32_t* qo_indptr, int32_t* q_offset,
    paged_kv_t<PageStorage::kIndices, QKVLayout::kNHD, half, int32_t> paged_kv, float* custom_mask,
    int32_t* qk_indptr, half* o, float* tmp, float* lse, uint32_t num_qo_tiles, float sm_scale,
    float rope_scale, float rope_theta, hipStream_t stream);

template hipError_t BatchPrefillWithPagedKVCacheDispatched<
  PageStorage::kIndices, QKVLayout::kNHD, 1, kPagesize,
  8, 256, PosEncodingMode::kNone, false, MaskMode::kCausal,
  half, half, int32_t>(
    half* q, int32_t* request_indices, int32_t* tile_indices, int32_t* qo_indptr, int32_t* q_offset,
    paged_kv_t<PageStorage::kIndices, QKVLayout::kNHD, half, int32_t> paged_kv, float* custom_mask,
    int32_t* qk_indptr, half* o, float* tmp, float* lse, uint32_t num_qo_tiles, float sm_scale,
    float rope_scale, float rope_theta, hipStream_t stream);

template hipError_t BatchPrefillWithPagedKVCacheDispatched<
  PageStorage::kIndices, QKVLayout::kNHD, 2, kPagesize,
  1, 64, PosEncodingMode::kNone, false, MaskMode::kCausal,
  half, half, int32_t>(
    half* q, int32_t* request_indices, int32_t* tile_indices, int32_t* qo_indptr, int32_t* q_offset,
    paged_kv_t<PageStorage::kIndices, QKVLayout::kNHD, half, int32_t> paged_kv, float* custom_mask,
    int32_t* qk_indptr, half* o, float* tmp, float* lse, uint32_t num_qo_tiles, float sm_scale,
    float rope_scale, float rope_theta, hipStream_t stream);

template hipError_t BatchPrefillWithPagedKVCacheDispatched<
  PageStorage::kIndices, QKVLayout::kNHD, 2, kPagesize,
  1, 128, PosEncodingMode::kNone, false, MaskMode::kCausal,
  half, half, int32_t>(
    half* q, int32_t* request_indices, int32_t* tile_indices, int32_t* qo_indptr, int32_t* q_offset,
    paged_kv_t<PageStorage::kIndices, QKVLayout::kNHD, half, int32_t> paged_kv, float* custom_mask,
    int32_t* qk_indptr, half* o, float* tmp, float* lse, uint32_t num_qo_tiles, float sm_scale,
    float rope_scale, float rope_theta, hipStream_t stream);

template hipError_t BatchPrefillWithPagedKVCacheDispatched<
  PageStorage::kIndices, QKVLayout::kNHD, 2, kPagesize,
  1, 256, PosEncodingMode::kNone, false, MaskMode::kCausal,
  half, half, int32_t>(
    half* q, int32_t* request_indices, int32_t* tile_indices, int32_t* qo_indptr, int32_t* q_offset,
    paged_kv_t<PageStorage::kIndices, QKVLayout::kNHD, half, int32_t> paged_kv, float* custom_mask,
    int32_t* qk_indptr, half* o, float* tmp, float* lse, uint32_t num_qo_tiles, float sm_scale,
    float rope_scale, float rope_theta, hipStream_t stream);

template hipError_t BatchPrefillWithPagedKVCacheDispatched<
  PageStorage::kIndices, QKVLayout::kNHD, 2, kPagesize,
  4, 64, PosEncodingMode::kNone, false, MaskMode::kCausal,
  half, half, int32_t>(
    half* q, int32_t* request_indices, int32_t* tile_indices, int32_t* qo_indptr, int32_t* q_offset,
    paged_kv_t<PageStorage::kIndices, QKVLayout::kNHD, half, int32_t> paged_kv, float* custom_mask,
    int32_t* qk_indptr, half* o, float* tmp, float* lse, uint32_t num_qo_tiles, float sm_scale,
    float rope_scale, float rope_theta, hipStream_t stream);

template hipError_t BatchPrefillWithPagedKVCacheDispatched<
  PageStorage::kIndices, QKVLayout::kNHD, 2, kPagesize,
  4, 128, PosEncodingMode::kNone, false, MaskMode::kCausal,
  half, half, int32_t>(
    half* q, int32_t* request_indices, int32_t* tile_indices, int32_t* qo_indptr, int32_t* q_offset,
    paged_kv_t<PageStorage::kIndices, QKVLayout::kNHD, half, int32_t> paged_kv, float* custom_mask,
    int32_t* qk_indptr, half* o, float* tmp, float* lse, uint32_t num_qo_tiles, float sm_scale,
    float rope_scale, float rope_theta, hipStream_t stream);

template hipError_t BatchPrefillWithPagedKVCacheDispatched<
  PageStorage::kIndices, QKVLayout::kNHD, 2, kPagesize,
  4, 256, PosEncodingMode::kNone, false, MaskMode::kCausal,
  half, half, int32_t>(
    half* q, int32_t* request_indices, int32_t* tile_indices, int32_t* qo_indptr, int32_t* q_offset,
    paged_kv_t<PageStorage::kIndices, QKVLayout::kNHD, half, int32_t> paged_kv, float* custom_mask,
    int32_t* qk_indptr, half* o, float* tmp, float* lse, uint32_t num_qo_tiles, float sm_scale,
    float rope_scale, float rope_theta, hipStream_t stream);

template hipError_t BatchPrefillWithPagedKVCacheDispatched<
  PageStorage::kIndices, QKVLayout::kNHD, 2, kPagesize,
  8, 64, PosEncodingMode::kNone, false, MaskMode::kCausal,
  half, half, int32_t>(
    half* q, int32_t* request_indices, int32_t* tile_indices, int32_t* qo_indptr, int32_t* q_offset,
    paged_kv_t<PageStorage::kIndices, QKVLayout::kNHD, half, int32_t> paged_kv, float* custom_mask,
    int32_t* qk_indptr, half* o, float* tmp, float* lse, uint32_t num_qo_tiles, float sm_scale,
    float rope_scale, float rope_theta, hipStream_t stream);

template hipError_t BatchPrefillWithPagedKVCacheDispatched<
  PageStorage::kIndices, QKVLayout::kNHD, 2, kPagesize,
  8, 128, PosEncodingMode::kNone, false, MaskMode::kCausal,
  half, half, int32_t>(
    half* q, int32_t* request_indices, int32_t* tile_indices, int32_t* qo_indptr, int32_t* q_offset,
    paged_kv_t<PageStorage::kIndices, QKVLayout::kNHD, half, int32_t> paged_kv, float* custom_mask,
    int32_t* qk_indptr, half* o, float* tmp, float* lse, uint32_t num_qo_tiles, float sm_scale,
    float rope_scale, float rope_theta, hipStream_t stream);

template hipError_t BatchPrefillWithPagedKVCacheDispatched<
  PageStorage::kIndices, QKVLayout::kNHD, 2, kPagesize,
  8, 256, PosEncodingMode::kNone, false, MaskMode::kCausal,
  half, half, int32_t>(
    half* q, int32_t* request_indices, int32_t* tile_indices, int32_t* qo_indptr, int32_t* q_offset,
    paged_kv_t<PageStorage::kIndices, QKVLayout::kNHD, half, int32_t> paged_kv, float* custom_mask,
    int32_t* qk_indptr, half* o, float* tmp, float* lse, uint32_t num_qo_tiles, float sm_scale,
    float rope_scale, float rope_theta, hipStream_t stream);
} // namespace flashinfer
