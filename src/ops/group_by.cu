#include "hip/hip_runtime.h"
/* Copyright 2019 Stanford
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "model.h"
#include "cuda_helper.h"
#include <math.h>
#include <stdio.h>
//#include "moe.h"

#define MAX_K 4
#define MAX_N 16
#define MAX_BATCH_SIZE 250


// group_by with local capacity factors
void FFModel::group_by(const Tensor& input,
                        const Tensor& assign,
                        Tensor* outputs,
                        int n, std::vector<float> alpha,
                        const char* name)
{
  GroupBy* group_by = new GroupBy(*this, input, assign, n, alpha, true, name);
  layers.push_back(group_by);
  for (int i = 0; i < n; i++)
    outputs[i] = group_by->outputs[i];
}

// group_by with global capacity factors
void FFModel::group_by(const Tensor& input,
                        const Tensor& assign,
                        Tensor* outputs,
                        int n, float alpha,
                        const char* name)
{
  std::vector<float> alpha_vec = {alpha};
  GroupBy* group_by = new GroupBy(*this, input, assign, n, alpha_vec, false, name);
  layers.push_back(group_by);
  for (int i = 0; i < n; i++)
    outputs[i] = group_by->outputs[i];
}


GroupBy::GroupBy(FFModel& model,
                  const Tensor& _input,
                  const Tensor& _assign,
                  int _n, std::vector<float> _alpha,
                  bool _local_lambda,
                  const char* name)
: Op(model, OP_GROUP_BY, name, _input, _assign),
  n(_n), local_lambda(_local_lambda),
  profiling(model.config.profiling)
{
  // TODO: Geht eleganter?
  for(size_t i = 0; i < _alpha.size(); i++) alpha.push_back(_alpha[i]);

  first_init = true;

  // FIXME: For now, set upper limits Better: Do as follows, but memory is
  // assigned per block, so requires to check that
  // https://stackoverflow.com/questions/5531247/allocating-shared-memory/5531640#5531640
  assert(n <= MAX_N && "Increase MAX_N in #define");
  assert(inputs[1].adim[0] <= MAX_K && "Increase MAX_K in #define");
  assert(inputs[0].adim[1] <= MAX_BATCH_SIZE && "Increase MAX_BATCH_SIZE in #define");

  int num_dim = _input.numDim;
  assert(_assign.numDim == 2);
  assert(_input.adim[num_dim-1] == _assign.adim[1]);
  assert(n > 0);

  // output dims
  int k = _assign.adim[0];
  int batch_size = inputs[1].adim[1];
  for(int i = 0; i < n; i++) {
    outputs[i].numDim = num_dim;
    for(int j = 0; j < num_dim-1; j++) {
      outputs[i].adim[j] = inputs[0].adim[j];
    }
    if(local_lambda) {
      outputs[i].adim[num_dim-1] = (int)ceil(alpha[i]*k/n*batch_size);
    } else {
      outputs[i].adim[num_dim-1] = (int)ceil(alpha[0]*k/n*batch_size);
    }
  }

  numWeights = 0;
  numOutputs = n;
}


void GroupBy::create_weights(FFModel& model)
{
  // Do nothing
}

template<int NDIM>
void GroupBy::create_output_and_partition_with_dim(FFModel& model)
{
  // Retrieve the task indexspace for the op
  std::string pcname = name;
  task_is = IndexSpaceT<NDIM>(model.get_or_create_task_is(NDIM, pcname));
  Context ctx = model.config.lg_ctx;
  Runtime* runtime = model.config.lg_hlr;
  Rect<NDIM> part_rect = runtime->get_index_space_domain(ctx, task_is);

  // Can only partition over the sample dim
  assert(part_rect.hi[0] == part_rect.lo[0]);

  int k = inputs[1].adim[0];
  int dims[NDIM];
  dims[0] = (int)ceil(alpha[0]*k/n*inputs[1].adim[1]);
  for(int i = 1; i < NDIM; i++) {
    dims[i] = inputs[0].adim[NDIM-i-1];
  }
  for(int i = 0; i < n; i++) {
    if(local_lambda)
      dims[0] = (int)ceil(alpha[i]*k/n*inputs[1].adim[1]);
    outputs[i] = model.create_tensor<NDIM>(dims, inputs[0].data_type, this);
    outputs[i].owner_op = this;
    outputs[i].owner_idx = i;
  }

  // Compute partition bound for input
  model.create_data_parallel_partition_with_diff_dims<2, NDIM>(
      inputs[1], (IndexSpaceT<NDIM>)task_is, input_lps[1], input_grad_lps[1]);
  Rect<NDIM> input_rect = runtime->get_index_partition_color_space(
      ctx, inputs[0].part.get_index_partition());
  if (input_rect == part_rect) {
    input_lps[0] = inputs[0].part;
    input_grad_lps[0] = inputs[0].part_grad;
  } else {
    model.create_disjoint_partition<NDIM>(
      inputs[0], (IndexSpaceT<NDIM>)task_is, input_lps[0], input_grad_lps[0]);
  }
}


void GroupBy::create_output_and_partition(FFModel& model)
{
  int dim = inputs[0].numDim;
  switch (dim) {
#define DIMFUNC(DIM) \
    case DIM: \
    { \
      create_output_and_partition_with_dim<DIM>(model); \
      break; \
    }
    LEGION_FOREACH_N(DIMFUNC)
#undef DIMFUNC
    default:
    {
      // Unsupported dim for ElementWiseBinary operator
      assert(false);
    }
  }
}

OpMeta* GroupBy::init_task(const Task* task,
                        const std::vector<PhysicalRegion> &regions,
                        Context ctx, Runtime* runtime)
{
  GroupBy* gb = (GroupBy*) task->args;
  FFHandler handle = *((FFHandler*)task->local_args);
  GroupByMeta* m = new GroupByMeta(handle, gb->n, gb->local_lambda);
  m->profiling = gb->profiling;
  /* NOTE: if this is the first time GroupBy is initalized (not a recompile),
  init the score with alpha */
  if(gb->first_init) {
    int copy_size = gb->local_lambda ? gb->n : 1;
    hipMemcpy(m->score, &gb->alpha[0], copy_size*sizeof(float), hipMemcpyHostToDevice);
  }
  gb->first_init = false;
  return m;
}


void GroupBy::init(const FFModel& ff)
{
  ArgumentMap argmap;
  Context ctx = ff.config.lg_ctx;
  Runtime* runtime = ff.config.lg_hlr;
  Domain domain = runtime->get_index_space_domain(ctx, task_is);
  switch (domain.get_dim()) {
#define DIMFUNC(DIM) \
    case DIM: \
    { \
      Rect<DIM> rect = domain; \
      ParallelConfig pc; \
      std::string pcname = name; \
      ff.config.find_parallel_config(DIM, pcname, pc); \
      int idx = 0; \
      for (PointInRectIterator<DIM> it(rect); it(); it++) { \
        FFHandler handle = ff.handlers[pc.device_ids[idx++]]; \
        argmap.set_point(*it, TaskArgument(&handle, sizeof(FFHandler))); \
      } \
      break; \
    }
    LEGION_FOREACH_N(DIMFUNC)
#undef DIMFUNC
    default:
      assert(false);
  }

  IndexLauncher launcher(GROUP_BY_INIT_TASK_ID, task_is,
                         TaskArgument(this, sizeof(GroupBy)), argmap,
                         Predicate::TRUE_PRED, false/*must*/, 0/*mapper_id*/,
                         FFConfig::get_hash_id(std::string(name)));
  // // data
  // launcher.add_region_requirement(
  //   RegionRequirement(input_lps[0], 0/*projection id*/,
  //     READ_ONLY, EXCLUSIVE, inputs[0].region));
  // launcher.add_field(0, FID_DATA);
  // // assign
  // launcher.add_region_requirement(
  //   RegionRequirement(input_lps[1], 0/*projection id*/,
  //     READ_ONLY, EXCLUSIVE, inputs[1].region));
  // launcher.add_field(1, FID_DATA);
  // output
  for(int i = 0; i < n; i++) {
    launcher.add_region_requirement(
      RegionRequirement(outputs[i].part, 0/*projection id*/,
        WRITE_ONLY, EXCLUSIVE, outputs[i].region));
    launcher.add_field(i, FID_DATA);
  }
  FutureMap fm = runtime->execute_index_space(ctx, launcher);
  fm.wait_all_results();
  switch (domain.get_dim()) {
#define DIMFUNC(DIM) \
    case DIM: \
    { \
      Rect<DIM> rect = domain; \
      int idx = 0; \
      for (PointInRectIterator<DIM> it(rect); it(); it++) { \
        meta[idx++] = fm.get_result<OpMeta*>(*it); \
      } \
      break; \
    }
    LEGION_FOREACH_N(DIMFUNC)
#undef DIMFUNC
    default:
      assert(false);
  }
}


__global__
void gb_forward_kernel(const float* input,
        const int* exp_assign,
        float** outputs,
        int n, // num experts
        int k, // chosen experts
        float* alpha, // factor additional memory assigned
        int batch_size,
        int data_dim,
        float* score,
        const bool local_lambda)
{
  __shared__ float* chosen_exp_preds[MAX_K*MAX_BATCH_SIZE];

  // Get pred pointers, single thread per block
  if(threadIdx.x == 0) {
    int exp_tensor_rows = ceil(alpha[0]*k/n*batch_size);
    int expert_idx[MAX_N] = {0};
    for(int i = 0; i < k*batch_size; i++) {
      // Get pointer to chosen expert predictions
      int expert = exp_assign[i];
      if(local_lambda) exp_tensor_rows = ceil(alpha[expert]*k/n*batch_size);
      if(expert_idx[expert] >= exp_tensor_rows) {
        // dropped sample
        chosen_exp_preds[i] = 0;
      }
      else {
        float* out = outputs[expert];
        chosen_exp_preds[i] = out + expert_idx[expert]*data_dim;
      }
      expert_idx[expert]++;
    }

    // compute score: min alpha such that all samples fit
    // TODO: You could do that in parallel with compute output
    float fact = 0.01f;
    float fact_1 = 1.0f - fact;
    float norm = (float)n/(k*batch_size)*fact;
    if(local_lambda) {
      for(int i = 0; i < n; i++) {
        score[i] = fact_1*score[i] + norm*expert_idx[i];
      }
    } else {
      float min_alpha = -1.0f;
      for(int i = 0; i < n; i++)
        if(expert_idx[i] > min_alpha)
          min_alpha = expert_idx[i];
      score[0] = fact_1*(*score) + norm*min_alpha;
    }
  }

  __syncthreads();

  // compute output
  CUDA_KERNEL_LOOP(i, k*batch_size*data_dim)
  {
    if(chosen_exp_preds[i/data_dim] != 0) {
      float a = input[(i/(k*data_dim))*data_dim + i%data_dim];
      chosen_exp_preds[i/data_dim][i%data_dim] = a;
    }
  }
}


// __global__
// void gb_backward_kernel(float* input_grad,
//         const int* exp_assign,
//         float** output_grads,
//         int n, // num experts
//         int k, // chosen experts
//         float alpha, // factor additional memory assigned
//         int batch_size,
//         int data_dim)
// {
//   __shared__ float* chosen_exp_grads[MAX_K*MAX_BATCH_SIZE];
//
//   // Get pred pointers, single thread
//   if(blockIdx.x * blockDim.x + threadIdx.x == 0) {
//     int exp_tensor_rows = ceil(alpha*k/n*batch_size);
//     int expert_idx[MAX_N] = {0};
//     for(int i = 0; i < k*batch_size; i++) {
//       // Get pointer to chosen expert predictions
//       int expert = exp_assign[i];
//       if(expert_idx[expert] >= exp_tensor_rows) {
//         // dropped sample
//         chosen_exp_grads[i] = 0;
//         continue;
//       }
//       chosen_exp_grads[i] = output_grads[expert] + expert_idx[expert]*data_dim;
//       expert_idx[expert]++;
//     }
//   }
//
//   __syncthreads();
//
//   // compute output
//   CUDA_KERNEL_LOOP(i, k*batch_size*data_dim)
//   {
//     if(chosen_exp_grads[i/data_dim] != 0) {
//       input_grad[(i/(k*data_dim))*data_dim + i%data_dim] = chosen_exp_grads[i/data_dim][i%data_dim];
//     }
//   }
// }

template<int NDIM>
float* GroupBy::forward_task_with_dim(const Task *task,
                            const std::vector<PhysicalRegion>& regions,
                            Context ctx, Runtime* runtime)
{
  // printf("gb fwd task\n");

  // Get n, alpha
  const GroupBy* gb = (GroupBy*) task->args;
  int n = gb->n;
  std::vector<float> alpha = gb->alpha;
  const bool local_lambda = gb->local_lambda;

  assert((int)regions.size() == n+2);
  assert((int)task->regions.size() == n+2);

  const GroupByMeta* m = *((GroupByMeta**)task->local_args);

  // get input and assign regions
  const AccessorRO<float, NDIM> acc_input(regions[0], FID_DATA);
  const AccessorRO<int, 2> acc_assign(regions[1], FID_DATA);

  Rect<NDIM> rect_input = runtime->get_index_space_domain(
      ctx, task->regions[0].region.get_index_space());
  Rect<2> rect_assign = runtime->get_index_space_domain(
      ctx, task->regions[1].region.get_index_space());

  int batch_size = rect_assign.hi[1] - rect_assign.lo[1] + 1;
  int data_dim = rect_input.volume()/batch_size;
  assert(batch_size == rect_input.hi[NDIM-1] - rect_input.lo[NDIM-1] + 1);
  int k = rect_assign.hi[0] - rect_assign.lo[0] + 1;

  // get output
  float* outputs[n];
  //int exp_output_rows = (int)ceil(alpha*k/n*batch_size);
  for(int i = 0; i < n; i++) {
    Domain out_domain = runtime->get_index_space_domain(
      ctx, task->regions[i+2].region.get_index_space());
    outputs[i] = helperGetTensorPointerWO<float>(
      regions[i+2], task->regions[i+2], FID_DATA, ctx, runtime);

    //coord_t output_rows = out_domain.hi()[1] - out_domain.lo()[1] + 1;
    // coord_t output_cols = out_domain.hi()[0] - out_domain.lo()[0] + 1;
    //assert((int)output_rows == exp_output_rows);
    // assert(output_cols == input_cols);
  }

  hipStream_t stream;
  checkCUDA(get_legion_stream(&stream));

  // call forward kernel
  int copy_size = local_lambda ? n : 1;
  hipMemcpy(m->dev_region_ptrs, outputs, n*sizeof(float*), hipMemcpyHostToDevice);
  hipMemcpy(m->alpha_pass, &alpha[0], copy_size*sizeof(float), hipMemcpyHostToDevice);

  // TODO: several blocks
  gb_forward_kernel<<<1, min(CUDA_NUM_THREADS,(int)(batch_size*k*data_dim))>>>(
    acc_input.ptr(rect_input), acc_assign.ptr(rect_assign), m->dev_region_ptrs, n, k,
    m->alpha_pass, batch_size, data_dim, m->score, local_lambda);

  float* score_ptr = new float[copy_size];
  hipMemcpy(score_ptr, m->score, copy_size*sizeof(float), hipMemcpyDeviceToHost);
  return score_ptr;
}


float* GroupBy::forward_task(const Task *task,
                          const std::vector<PhysicalRegion> &regions,
                          Context ctx, Runtime *runtime)
{
  Domain in_domain = runtime->get_index_space_domain(
      ctx, task->regions[0].region.get_index_space());
  switch (in_domain.get_dim()) {
#define DIMFUNC(DIM) \
    case DIM: \
      return forward_task_with_dim<DIM>(task, regions, ctx, runtime);
    LEGION_FOREACH_N(DIMFUNC)
#undef DIMFUNC
    default:
      assert(false);
  }
  return 0;
}


// template<int NDIM>
// void GroupBy::backward_task_with_dim(const Task *task,
//                             const std::vector<PhysicalRegion>& regions,
//                             Context ctx, Runtime* runtime)
// {
//   // Get n, alpha
//   const GroupByMeta* m = *((GroupByMeta**)task->local_args);
//   const GroupBy* gb = (GroupBy*) task->args;
//   int n = gb->n;
//   float alpha = gb->alpha;
//
//   assert((int)regions.size() == n+2);
//   assert((int)task->regions.size() == n+2);
//
//   // get input and assign regions
//   const AccessorWO<float, NDIM> acc_input_grad(regions[0], FID_DATA);
//   const AccessorRO<int, 2> acc_assign(regions[1], FID_DATA);
//
//   Rect<NDIM> rect_input_grad = runtime->get_index_space_domain(
//       ctx, task->regions[0].region.get_index_space());
//   Rect<2> rect_assign = runtime->get_index_space_domain(
//       ctx, task->regions[1].region.get_index_space());
//
//   int exp_batch_size = rect_input_grad.hi[NDIM-1] - rect_input_grad.hi[NDIM-1] + 1;
//   int batch_size = rect_assign.hi[1] - rect_assign.lo[1] + 1;
//   int data_dim = rect_input_grad.volume()/exp_batch_size;
//   int k = rect_assign.hi[0] - rect_assign.lo[0] + 1;
//
//   // get output
//   float* output_grads[n];
//   //int exp_output_rows = (int)ceil(alpha*k/n*batch_size);
//   for(int i = 0; i < n; i++) {
//     Domain out_domain = runtime->get_index_space_domain(
//       ctx, task->regions[i+2].region.get_index_space());
//     output_grads[i] = helperGetTensorPointerRW<float>(
//       regions[i+2], task->regions[i+2], FID_DATA, ctx, runtime);
//
//     // //coord_t output_rows = out_domain.hi()[1] - out_domain.lo()[1] + 1;
//     // coord_t output_cols = out_domain.hi()[0] - out_domain.lo()[0] + 1;
//     // //assert((int)output_rows == exp_output_rows);
//     // assert(output_cols == input_cols);
//   }
//
// // TODO: why cublas/cudnn stream is needed here
//   hipStream_t stream;
//   checkCUDA(get_legion_stream(&stream));
//
//   // call forward kernel
//   hipMemcpy(m->dev_region_ptrs, output_grads, n*sizeof(float*), hipMemcpyHostToDevice);
//
//   gb_backward_kernel<<<GET_BLOCKS(batch_size*k*data_dim), min(CUDA_NUM_THREADS,(int)(batch_size*k*data_dim)), 0, stream>>>(
//     acc_input_grad.ptr(rect_input_grad), acc_assign.ptr(rect_assign), m->dev_region_ptrs,
//     n, k, alpha, batch_size, data_dim);
// }

void GroupBy::backward_task(const Task *task,
                          const std::vector<PhysicalRegion> &regions,
                          Context ctx, Runtime *runtime)
{
  return;
//   Domain in_domain = runtime->get_index_space_domain(
//       ctx, task->regions[0].region.get_index_space());
//   switch (in_domain.get_dim()) {
// #define DIMFUNC(DIM) \
//     case DIM: \
//       return backward_task_with_dim<DIM>(task, regions, ctx, runtime);
//     LEGION_FOREACH_N(DIMFUNC)
// #undef DIMFUNC
//     default:
//       assert(false);
//   }
}


void GroupBy::forward(const FFModel& ff)
{
  ArgumentMap argmap;
  Context ctx = ff.config.lg_ctx;
  Runtime* runtime = ff.config.lg_hlr;
  Domain domain = runtime->get_index_space_domain(ctx, task_is);
  switch (domain.get_dim()) {
#define DIMFUNC(DIM) \
    case DIM: \
    { \
      Rect<DIM> rect = domain; \
      int idx = 0; \
      for (PointInRectIterator<DIM> it(rect); it(); it++) { \
        OpMeta* mp = meta[idx++]; \
        argmap.set_point(*it, TaskArgument(&mp, sizeof(OpMeta*))); \
      } \
      break; \
    }
    LEGION_FOREACH_N(DIMFUNC)
#undef DIMFUNC
    default:
      assert(false);
  }
  IndexLauncher launcher(GROUP_BY_FWD_TASK_ID, task_is,
                         TaskArgument(this, sizeof(GroupBy)), argmap,
                         Predicate::TRUE_PRED, false/*must*/, 0/*mapper_id*/,
                         FFConfig::get_hash_id(std::string(name)));
  // data
  launcher.add_region_requirement(
    RegionRequirement(input_lps[0], 0/*projection id*/,
      READ_ONLY, EXCLUSIVE, inputs[0].region));
  launcher.add_field(0, FID_DATA);

  // assign
  launcher.add_region_requirement(
    RegionRequirement(input_lps[1], 0/*projection id*/,
      READ_ONLY, EXCLUSIVE, inputs[1].region));
  launcher.add_field(1, FID_DATA);

  // output
  for(int i = 0; i < n; i++) {
    launcher.add_region_requirement(
      RegionRequirement(outputs[i].part, 0/*projection id*/,
        WRITE_ONLY, EXCLUSIVE, outputs[i].region));
    launcher.add_field(i+2, FID_DATA);
  }
  // printf("groupby launches!!\n");

  FutureMap score_fm = runtime->execute_index_space(ctx, launcher);
  // add score futures to GroupBy future vector attribute
 //TODO: Comment in for score
  switch (domain.get_dim()) {
#define DIMFUNC(DIM) \
    case DIM: \
    { \
      Rect<DIM> rect = domain; \
      for (PointInRectIterator<DIM> it(rect); it(); it++) \
        score_futures.push_back(score_fm[*it]); \
      break; \
    }
    LEGION_FOREACH_N(DIMFUNC)
#undef DIMFUNC
    default:
      assert(false);
  }

}

void GroupBy::backward(const FFModel& ff)
{
  // TODO: That GroupBy needs to propagate gradients is unusual.
  // We could check if needs to be propagated and only do if inputs[0] is
  // the output of anyother operator. Else, don't propagate.
  // TODO: backward_task only supports 2D input for now.

  ArgumentMap argmap;
  Context ctx = ff.config.lg_ctx;
  Runtime* runtime = ff.config.lg_hlr;
  Domain domain = runtime->get_index_space_domain(ctx, task_is);
  switch (domain.get_dim()) {
#define DIMFUNC(DIM) \
    case DIM: \
    { \
      Rect<DIM> rect = domain; \
      int idx = 0; \
      for (PointInRectIterator<DIM> it(rect); it(); it++) { \
        OpMeta* mp = meta[idx++]; \
        argmap.set_point(*it, TaskArgument(&mp, sizeof(OpMeta*))); \
      } \
      break; \
    }
    LEGION_FOREACH_N(DIMFUNC)
#undef DIMFUNC
    default:
      assert(false);
  }
  IndexLauncher launcher(GROUP_BY_BWD_TASK_ID, task_is,
                         TaskArgument(this, sizeof(GroupBy)), argmap,
                         Predicate::TRUE_PRED, false/*must*/, 0/*mapper_id*/,
                         FFConfig::get_hash_id(std::string(name)));

  // input_grad
  launcher.add_region_requirement(
    RegionRequirement(input_grad_lps[0], 0/*projection id*/,
      WRITE_ONLY, EXCLUSIVE, inputs[0].region_grad));
  launcher.add_field(0, FID_DATA);

  // assign
  launcher.add_region_requirement(
    RegionRequirement(input_lps[1], 0/*projection id*/,
      READ_ONLY, EXCLUSIVE, inputs[1].region));
  launcher.add_field(1, FID_DATA);

  // output grad
  for(int i = 0; i < n; i++) {
    launcher.add_region_requirement(
      RegionRequirement(outputs[i].part_grad, 0/*projection id*/,
        WRITE_ONLY, EXCLUSIVE, outputs[i].region_grad));
    launcher.add_field(i+2, FID_DATA);
  }

  runtime->execute_index_space(ctx, launcher);
}


GroupByMeta::GroupByMeta(FFHandler handler, int n, bool local_lambda)
: OpMeta(handler)
{
  checkCUDA(hipMalloc(&dev_region_ptrs, n*sizeof(float*)));
  int copy_size = local_lambda ? n : 1;
  checkCUDA(hipMalloc(&score, copy_size*sizeof(float)));
  checkCUDA(hipMalloc(&alpha_pass, copy_size*sizeof(float)));
}
GroupByMeta::~GroupByMeta(void)
{
  checkCUDA(hipFree(&dev_region_ptrs));
  checkCUDA(hipFree(&score));
  checkCUDA(hipFree(&alpha_pass));
}


bool GroupBy::measure_operator_cost(Simulator* sim,
                                 const ParallelConfig& pc,
                                 CostMetrics& cost_metrics)
{
  //TODO: implement
  cost_metrics.forward_time = 0.0f;
  cost_metrics.backward_time = 0.0f;
  cost_metrics.memory_requirement = 0;
  return true;
}
