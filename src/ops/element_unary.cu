#include "hip/hip_runtime.h"
/* Copyright 2020 Stanford
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "flexflow/ops/element_unary.h"
#include "flexflow/utils/cuda_helper.h"

namespace FlexFlow {

// declare Legion names
using Legion::Context;
using Legion::Runtime;
using Legion::Domain;
using Legion::Task;
using Legion::Rect;
using Legion::PhysicalRegion;
using Legion::coord_t;

bool ElementUnary::can_inplace_output(void)
{
  return true;
}

bool ElementUnary::has_inplace_output(void)
{
  return inplace;
}

void ElementUnary::do_inplace_output(void)
{
  inplace = true;
}

bool ElementUnary::use_cudnn(OperatorType type)
{
  if (type == OP_RELU)
    return true;
  if (type == OP_SIGMOID)
    return true;
  if (type == OP_TANH)
    return true;
  if (type == OP_ELU)
    return true;
  return false;
}

OpMeta* ElementUnary::init_task(const Task *task,
                                const std::vector<PhysicalRegion> &regions,
                                Context ctx, Runtime *runtime)
{
  ElementUnary* eu = (ElementUnary*) task->args;
  FFHandler handle = *((FFHandler*) task->local_args);
  ElementUnaryMeta* m = new ElementUnaryMeta(handle);
  m->op_type = eu->op_type;
  m->data_type = eu->outputs[0]->data_type;
  // Current assume input and output have the same data type
  assert(eu->outputs[0]->data_type == eu->inputs[0]->data_type);
  m->profiling = eu->profiling;
  m->inplace = eu->inplace;
  m->scalar = eu->scalar;
  if (m->inplace) {
    assert(regions.size() == 1);
    assert(task->regions.size() == 1);
  } else {
    assert(regions.size() == 2);
    assert(task->regions.size() == 2);
  }

  if (use_cudnn(m->op_type))
  {
    hipdnnActivationMode_t mode;
    switch (m->op_type) {
      case OP_SIGMOID:
        mode = HIPDNN_ACTIVATION_SIGMOID;
        break;
      case OP_RELU:
        mode = HIPDNN_ACTIVATION_RELU;
        break;
      case OP_TANH:
        mode = HIPDNN_ACTIVATION_TANH;
        break;
      case OP_ELU:
        mode = HIPDNN_ACTIVATION_ELU;
        break;
      default:
        assert(false);
    }
    checkCUDNN(hipdnnSetActivationDescriptor(m->actiDesc, mode,
                                            HIPDNN_PROPAGATE_NAN, 0.0));
    Domain input_domain = runtime->get_index_space_domain(
        ctx, task->regions[0].region.get_index_space());
    checkCUDNN(cudnnSetTensorDescriptorFromDomain(m->inputTensor, input_domain));
    checkCUDNN(cudnnSetTensorDescriptorFromDomain(m->outputTensor, input_domain));
  }
  return m;
}

template<typename T>
__global__
void elewise_unary_forward_kernel(coord_t volume,
                                  const T scalar,
                                  OperatorType type,
                                  const T* in,
                                  T* out)
{
  CUDA_KERNEL_LOOP(i, volume)
  {
    switch (type) {
      case OP_EXP:
      {
        out[i] = (T) exp((float)in[i]);
        break;
      }
      case OP_IDENTITY:
      {
	out[i] = in[i];
	break;
      }
      case OP_SCALAR_MULTIPLY:
      {
	out[i] = in[i] * scalar;
	break;
      }
      case OP_SCALAR_ADD:
      {
	out[i] = in[i] + scalar;
	break;
      }
      case OP_SCALAR_SUB:
      {
	out[i] = in[i] - scalar;
	break;
      }
      case OP_SCALAR_TRUE_DIV:
      {
	out[i] = in[i] / scalar;
	break;
      }
      case OP_GELU:
      {
	out[i] = (T)(in[i] * 0.5 * erfc(-in[i]*M_SQRT1_2));
	break;
      }
      case OP_RSQRT:
      {
        out[i] = (T)(1.0f / sqrt((float)in[i]));
	break;
      }
      case OP_POW:
      {
        out[i] = (T)(powf(in[i], scalar));
        break;
      }
      default:
        assert(false);
    }
  }
}

/*static*/
template<typename T>
void ElementUnary::forward_kernel(const ElementUnaryMeta* m,
                                  const T* input_ptr,
                                  T* output_ptr,
                                  size_t num_elements, 
                                  hipStream_t stream)
{
  checkCUDNN(hipdnnSetStream(m->handle.dnn, stream));

  if (use_cudnn(m->op_type)) {
    float alpha = 1.0f, beta = 0.0f;
    checkCUDNN(hipdnnActivationForward(m->handle.dnn, m->actiDesc,
        &alpha, m->inputTensor, input_ptr,
        &beta, m->outputTensor, output_ptr));
  } else {
    elewise_unary_forward_kernel<<<GET_BLOCKS(num_elements), CUDA_NUM_THREADS, 0, stream>>>(
        num_elements, (T)m->scalar, m->op_type, input_ptr, output_ptr);
  }
}

void ElementUnary::forward_task(
    const Task* task,
    const std::vector<PhysicalRegion> &regions,
    Context ctx, Runtime* runtime)
{
  const ElementUnaryMeta* m = *((ElementUnaryMeta**) task->local_args);
  if (m->data_type == DT_FLOAT) {
    forward_task_with_type<float>(task, regions, ctx, runtime);
  } else if (m->data_type == DT_DOUBLE) {
    forward_task_with_type<double>(task, regions, ctx, runtime);
  } else if (m->data_type == DT_INT32) {
    forward_task_with_type<int32_t>(task, regions, ctx, runtime);
  } else if (m->data_type == DT_INT64) {
    forward_task_with_type<int64_t>(task, regions, ctx, runtime);
  } else {
    assert(false && "Unsupported data type in Embedding forward");
  }
}

/*
  regions[0](I): input
  regions[1](O): output
*/
template<typename DT>
void ElementUnary::forward_task_with_type(
    const Task* task,
    const std::vector<PhysicalRegion> &regions,
    Context ctx, Runtime* runtime)
{
  //const ElementUnary* ele = (const ElementUnary*) task->args;
  const ElementUnaryMeta* m = *((ElementUnaryMeta**) task->local_args);
  Domain input_domain = runtime->get_index_space_domain(
    ctx, task->regions[0].region.get_index_space());
  const DT* input_ptr = NULL;
  DT* output_ptr = NULL;
  if (m->inplace) {
    assert(regions.size() == 1);
    assert(task->regions.size() == 1);
    output_ptr = helperGetTensorPointerRW<DT>(
      regions[0], task->regions[0], FID_DATA, ctx, runtime);
    input_ptr = output_ptr;
  } else {
    assert(regions.size() == 2);
    assert(task->regions.size() == 2);
    Domain output_domain = runtime->get_index_space_domain(
      ctx, task->regions[1].region.get_index_space());
    assert(output_domain == input_domain);
    input_ptr = helperGetTensorPointerRO<DT>(
      regions[0], task->regions[0], FID_DATA, ctx, runtime);
    output_ptr = helperGetTensorPointerWO<DT>(
      regions[1], task->regions[1], FID_DATA, ctx, runtime);
  }

  hipStream_t stream;
  checkCUDA(get_legion_stream(&stream));
  forward_kernel(m, input_ptr, output_ptr, input_domain.get_volume(), stream);
}

template<typename T>
__global__
void elewise_unary_backward_kernel(coord_t volume,
				   const T scalar,
                                   OperatorType type,
                                   const T* output,
                                   const T* output_grad,
                                   const T* input,
                                   T* input_grad)
{
  CUDA_KERNEL_LOOP(i, volume)
  {
    switch (type) {
      case OP_EXP:
      {
        //TODO: change to use output instead of recomputing
        input_grad[i] += (T)(output_grad[i] * exp((float)input[i]));
        break;
      }
      case OP_IDENTITY:
      {
	input_grad[i] += output_grad[i];
	break;
      } 
      case OP_SCALAR_MULTIPLY:
      {
	input_grad[i] += output_grad[i]*scalar;
	break;
      }
      case OP_SCALAR_ADD:
      {
	input_grad[i] += output_grad[i];
	break;
      }
      case OP_SCALAR_SUB:
      {
	input_grad[i] += output_grad[i];
	break;
      }
      case OP_SCALAR_TRUE_DIV:
      {
	input_grad[i] += output_grad[i]/scalar;
	break;
      }
      case OP_GELU:
      {
	input_grad[i] = (T)(output_grad[i]*(0.5 * erfc(-input[i]*M_SQRT1_2)-0.5*M_SQRT1_2*input[i]*exp(-input[i]*input[i]*0.5)));
	break;
      }
      case OP_RSQRT:
      {
        input_grad[i] = (T)(-0.5f * output_grad[i] * output[i] * output[i] * output[i]);
	break;
      }
      case OP_POW:
      {
        input_grad[i] = (T)(output_grad[i] * scalar * powf(input[i], scalar - 1));
      }
      default:
        assert(false);
    }
  }
}

/*static*/
template<typename DT>
void ElementUnary::backward_kernel(const ElementUnaryMeta* m,
                                   const DT* input_ptr,
                                   DT* input_grad_ptr,
                                   const DT* output_ptr,
                                   const DT* output_grad_ptr,
                                   size_t num_elements,
                                   hipStream_t stream)
{
  checkCUDNN(hipdnnSetStream(m->handle.dnn, stream));

  if (use_cudnn(m->op_type)) {
    float alpha = 1.0f;
    checkCUDNN(hipdnnActivationBackward(m->handle.dnn, m->actiDesc,
        &alpha, m->outputTensor, output_ptr, m->outputTensor, output_grad_ptr,
        m->inputTensor, input_ptr, &alpha, m->inputTensor, input_grad_ptr));
  } else {
    elewise_unary_backward_kernel<DT><<<GET_BLOCKS(num_elements), CUDA_NUM_THREADS, 0, stream>>>(
        num_elements, m->scalar, m->op_type, output_ptr, output_grad_ptr, input_ptr, input_grad_ptr);
  }
}

void ElementUnary::backward_task(
    const Task* task,
    const std::vector<PhysicalRegion> &regions,
    Context ctx, Runtime* runtime)
{
  const ElementUnaryMeta* m = *((ElementUnaryMeta**) task->local_args);
  if (m->data_type == DT_FLOAT) {
    backward_task_with_type<float>(task, regions, ctx, runtime);
  } else if (m->data_type == DT_DOUBLE) {
    backward_task_with_type<double>(task, regions, ctx, runtime);
  } else if (m->data_type == DT_INT32) {
    backward_task_with_type<int32_t>(task, regions, ctx, runtime);
  } else if (m->data_type == DT_INT64) {
    backward_task_with_type<int64_t>(task, regions, ctx, runtime);
  } else {
    assert(false && "Unsupported data type in Embedding forward");
  }
}

/*
  regions[0](I): input
  regions[1](I/O): input_grad
  regions[2](I): output
  regions[3](I): output_grad
*/
template<typename DT>
void ElementUnary::backward_task_with_type(
    const Task* task,
    const std::vector<PhysicalRegion> &regions,
    Context ctx, Runtime* runtime)
{
  //const ElementUnary* ele = (const ElementUnary*) task->args;
  const ElementUnaryMeta* m = *((ElementUnaryMeta**) task->local_args);
  const DT* input_ptr = NULL, *output_ptr = NULL, *output_grad_ptr = NULL;
  DT* input_grad_ptr = NULL;
  Domain input_domain = runtime->get_index_space_domain(
    ctx, task->regions[0].region.get_index_space());
  if (m->inplace) {
    assert(regions.size() == 2);
    assert(task->regions.size() == 2);
    Domain input_grad_domain = runtime->get_index_space_domain(
      ctx, task->regions[1].region.get_index_space());
    assert(input_grad_domain == input_domain);
    input_ptr = helperGetTensorPointerRO<DT>(
      regions[0], task->regions[0], FID_DATA, ctx, runtime);
    input_grad_ptr = helperGetTensorPointerRW<DT>(
      regions[1], task->regions[1], FID_DATA, ctx, runtime);
    output_ptr = input_ptr;
    output_grad_ptr = input_grad_ptr;
  } else {
    assert(regions.size() == 4);
    assert(task->regions.size() == 4);
    Domain input_grad_domain = runtime->get_index_space_domain(
      ctx, task->regions[1].region.get_index_space());
    Domain output_domain = runtime->get_index_space_domain(
      ctx, task->regions[2].region.get_index_space());
    Domain output_grad_domain = runtime->get_index_space_domain(
      ctx, task->regions[3].region.get_index_space());
    assert(output_grad_domain == input_domain);
    assert(output_grad_domain == output_domain);
    assert(output_grad_domain == input_grad_domain);
    input_ptr = helperGetTensorPointerRO<DT>(
      regions[0], task->regions[0], FID_DATA, ctx, runtime);
    input_grad_ptr = helperGetTensorPointerRW<DT>(
      regions[1], task->regions[1], FID_DATA, ctx, runtime);
    output_ptr = helperGetTensorPointerRO<DT>(
      regions[2], task->regions[2], FID_DATA, ctx, runtime);
    output_grad_ptr = helperGetTensorPointerRO<DT>(
      regions[3], task->regions[3], FID_DATA, ctx, runtime);
  }

  hipStream_t stream;
  checkCUDA(get_legion_stream(&stream));
  backward_kernel<DT>(m, input_ptr, input_grad_ptr, output_ptr, output_grad_ptr, input_domain.get_volume(), stream);
}

ElementUnaryMeta::ElementUnaryMeta(FFHandler handler)
: OpMeta(handler)
{
  checkCUDNN(hipdnnCreateTensorDescriptor(&inputTensor));
  checkCUDNN(hipdnnCreateTensorDescriptor(&outputTensor));
  checkCUDNN(hipdnnCreateActivationDescriptor(&actiDesc));
}

bool ElementUnary::measure_operator_cost(Simulator* sim,
                                         const ParallelConfig& pc,
                                         CostMetrics& cost_metrics) const
{
  ParallelTensorBase sub_output, sub_input;
  if (!outputs[0]->get_output_sub_tensor(pc, sub_output, op_type))
    return false;
  if (!inputs[0]->get_input_sub_tensor(pc, sub_input, op_type))
    return false;
  ElementUnaryMeta* m = sim->ele_unary_meta;
  m->op_type = op_type;
  if (use_cudnn(m->op_type))
  {
    hipdnnActivationMode_t mode;
    switch (op_type) {
      case OP_SIGMOID:
        mode = HIPDNN_ACTIVATION_SIGMOID;
        break;
      case OP_RELU:
        mode = HIPDNN_ACTIVATION_RELU;
        break;
      case OP_TANH:
        mode = HIPDNN_ACTIVATION_TANH;
        break;
      case OP_ELU:
        mode = HIPDNN_ACTIVATION_ELU;
        break;
      default:
        assert(false);
    }
    checkCUDNN(hipdnnSetActivationDescriptor(m->actiDesc, mode,
                                            HIPDNN_PROPAGATE_NAN, 0.0));
    Domain input_domain, output_domain;
    input_domain.dim = sub_input.num_dims;
    for (int i = 0; i < sub_input.num_dims; i++) {
      input_domain.rect_data[i] = 0;
      input_domain.rect_data[i+input_domain.dim] = sub_input.dims[i].size-1;
    }
    output_domain.dim = sub_output.num_dims;
    for (int i = 0; i < sub_output.num_dims; i++) {
      output_domain.rect_data[i] = 0;
      output_domain.rect_data[i+input_domain.dim] = sub_output.dims[i].size-1;
    }
    checkCUDNN(cudnnSetTensorDescriptorFromDomain(m->inputTensor, input_domain));
    checkCUDNN(cudnnSetTensorDescriptorFromDomain(m->outputTensor, output_domain));
  }
  sim->free_all();
  float* input_ptr = (float*)sim->allocate(sub_input.get_volume(), DT_FLOAT);
  assert(input_ptr != NULL);
  float* output_ptr = NULL;
  if (inplace) {
    output_ptr = input_ptr;
  } else {
    output_ptr = (float*)sim->allocate(sub_output.get_volume(), DT_FLOAT);
  }
  assert(output_ptr != NULL);

  hipStream_t stream;
  checkCUDA(get_legion_stream(&stream));
  std::function<void()> forward, backward;
  forward = [&] {
    forward_kernel(m, input_ptr, output_ptr, sub_output.get_volume(), stream);
  };
  if (sim->computationMode == COMP_MODE_TRAINING) {
    float* input_grad_ptr = (float*)sim->allocate(sub_input.get_volume(), DT_FLOAT);
    assert(input_grad_ptr != NULL);
    float* output_grad_ptr = NULL;
    if (inplace) {
      output_grad_ptr = input_grad_ptr;
    } else {
      output_grad_ptr = (float*)sim->allocate(sub_output.get_volume(), DT_FLOAT);
    }
    assert(output_grad_ptr != NULL);
    backward = [&] {
      backward_kernel(m, input_ptr, input_grad_ptr, output_ptr, output_grad_ptr,
          sub_output.get_volume(), stream);
    };
  }

  inner_measure_operator_cost(sim, forward, backward, cost_metrics);

  if (sim->computationMode == COMP_MODE_TRAINING) {
    log_measure.debug("[Measure Elewise Unary] name(%s) num_elements(%zu) forward_time(%.4lf) backward_time(%.4lf)\n",
        name, sub_output.get_volume(),
        cost_metrics.forward_time,
        cost_metrics.backward_time);
  } else {
    log_measure.debug("[Measure Elewise Unary] name(%s) num_elements(%zu) forward_time(%.4lf)\n",
        name, sub_output.get_volume(),
        cost_metrics.forward_time);
  }
  return true;
}

}; // namespace FlexFlow
