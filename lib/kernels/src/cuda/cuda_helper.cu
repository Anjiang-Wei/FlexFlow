#include "hip/hip_runtime.h"
#include "device.h"
#include "kernels/datatype_dispatch.h"

namespace FlexFlow {

#ifdef FF_USE_CUDA
hipError_t get_legion_stream(hipStream_t *stream) {
#ifdef DISABLE_LEGION_CUDA_HIJACK
  *stream = (hipStream_t)0;
  return hipSuccess;
#else
  return hipStreamCreate(stream);
#endif
}
#elif FF_USE_HIP_CUDA
extern "C" {
hipStream_t hipGetTaskStream();
}
hipError_t get_legion_stream(hipStream_t *stream) {
#ifdef DISABLE_LEGION_CUDA_HIJACK
  *stream = (hipStream_t)0;
#else
  *stream = hipGetTaskStream();
#endif
  return hipSuccess;
}
#else
#error "Unknown device, please make sure if CUDA is enabled"
#endif

}; // namespace FlexFlow

using FlexFlow::get_legion_stream;

__global__ void scale_kernel(float *ptr, coord_t size, float a, float b) {
  CUDA_KERNEL_LOOP(i, size) {
    ptr[i] = (b - a) * ptr[i] + a;
  }
}

__global__ void ones_kernel(float *ptr, coord_t size) {
  CUDA_KERNEL_LOOP(i, size) {
    ptr[i] = 1.0f;
  }
}

template <typename DT>
__global__ void assign_kernel(DT *ptr, size_t size, DT value) {
  CUDA_KERNEL_LOOP(i, size) {
    ptr[i] = value;
  }
}

template <typename DT>
__global__ void copy_kernel(DT *dst, const DT *src, coord_t size) {
  CUDA_KERNEL_LOOP(i, size) {
    dst[i] = src[i];
  }
}

template <typename DT>
__global__ void reluBackward(DT *grad_ptr, const DT *output, size_t n) {
  CUDA_KERNEL_LOOP(i, n) {
    grad_ptr[i] = (output[i] > 0.0f) ? grad_ptr[i] : 0;
  }
}

__host__ void relu_backward_kernel(DataType data_type,
                                   void *output_grad_ptr,
                                   void const *output_ptr,
                                   size_t output_size,
                                   hipStream_t stream) {
  if (data_type == DataType::FLOAT) {
    reluBackward<float>
        <<<GET_BLOCKS(output_size), CUDA_NUM_THREADS, 0, stream>>>(
            (float *)output_grad_ptr, (float const *)output_ptr, output_size);
  } else if (data_type == DataType::DOUBLE) {
    reluBackward<double>
        <<<GET_BLOCKS(output_size), CUDA_NUM_THREADS, 0, stream>>>(
            (double *)output_grad_ptr, (double const *)output_ptr, output_size);
  } else {
    assert(false && "Unsupported data type in Linear backward");
    exit(1);
  }
}

template <typename DT>
__global__ void
    sigmoid_backward_function(DT *grad_ptr, const DT *output, size_t n) {
  CUDA_KERNEL_LOOP(i, n) {
    grad_ptr[i] = grad_ptr[i] * output[i] * (1.0f - output[i]);
  }
}

__host__ void sigmoid_backward_kernel(DataType data_type,
                                      void *output_grad_ptr,
                                      void const *output_ptr,
                                      size_t output_size,
                                      hipStream_t stream) {
  if (data_type == DataType::FLOAT) {
    sigmoid_backward_function<float>
        <<<GET_BLOCKS(output_size), CUDA_NUM_THREADS, 0, stream>>>(
            (float *)output_grad_ptr, (float const *)output_ptr, output_size);
  } else if (data_type == DataType::DOUBLE) {
    sigmoid_backward_function<double>
        <<<GET_BLOCKS(output_size), CUDA_NUM_THREADS, 0, stream>>>(
            (double *)output_grad_ptr, (double const *)output_ptr, output_size);
  } else {
    assert(false && "Unsupported data type in Linear backward");
    exit(1);
  }
}

__global__ void gelu_forward_kernel(size_t size,
                                    float const B,
                                    float const C,
                                    float *input) {
  CUDA_KERNEL_LOOP(i, size) {
    float const in = input[i];
    float const cdf = 0.5f + 0.5f * tanh(in * (C * in * in + B));
    input[i] = in * cdf;
  }
}

__global__ void
    apply_add(float *data_ptr, float const *replica_ptr, size_t size) {
  CUDA_KERNEL_LOOP(i, size) {
    data_ptr[i] += replica_ptr[i];
  }
}

template <typename T>
__global__ void
    apply_add_with_scale(T *data_ptr, T const *grad_ptr, size_t size, T scale) {
  CUDA_KERNEL_LOOP(i, size) {
    data_ptr[i] += grad_ptr[i] * scale;
  }
}

template <typename T>
__global__ void add_kernel(T *data_ptr, T const *grad_ptr, size_t size) {
  CUDA_KERNEL_LOOP(i, size) {
    data_ptr[i] += grad_ptr[i];
  }
}

__global__ void add_with_stride(float *output,
                                float const *input,
                                int num_blocks,
                                int output_blk_size,
                                int input_blk_size) {
  int min_blk_size = min(output_blk_size, input_blk_size);
  CUDA_KERNEL_LOOP(i, num_blocks * min_blk_size) {
    int blk_idx = i / min_blk_size;
    int blk_offset = i % min_blk_size;
    int input_offset = blk_idx * input_blk_size + blk_offset;
    int output_offset = blk_idx * output_blk_size + blk_offset;
    output[output_offset] += input[input_offset];
  }
}

__global__ void copy_with_stride(float *output,
                                 float const *input,
                                 int num_blocks,
                                 int output_blk_size,
                                 int input_blk_size) {
  int min_blk_size = min(output_blk_size, input_blk_size);
  CUDA_KERNEL_LOOP(i, num_blocks * min_blk_size) {
    int blk_idx = i / min_blk_size;
    int blk_offset = i % min_blk_size;
    int input_offset = blk_idx * input_blk_size + blk_offset;
    int output_offset = blk_idx * output_blk_size + blk_offset;
    output[output_offset] = input[input_offset];
  }
}

__host__ void updateGAS(float *para_ptr,
                        float const *grad_ptr,
                        size_t replica_size,
                        int num_replica,
                        float learning_rate) {
  hipStream_t stream;
  checkCUDA(get_legion_stream(&stream));
  // Step 1: gater gradients to the first replica
  for (int i = 1; i < num_replica; i++) {
    float const *replica = grad_ptr + i * replica_size;
    apply_add<<<GET_BLOCKS(replica_size), CUDA_NUM_THREADS, 0, stream>>>(
        (float *)grad_ptr, replica, replica_size);
  }
  // Step 2: scale the first replica
  float scale_factor = 1.0f / num_replica * (-learning_rate);
  apply_add_with_scale<<<GET_BLOCKS(replica_size),
                         CUDA_NUM_THREADS,
                         0,
                         stream>>>(
      para_ptr, grad_ptr, replica_size, scale_factor);
}

template <typename T>
__host__ void
    print_tensor(T const *ptr, size_t num_elements, char const *prefix) {
  // device synchronize to make sure the data are ready
  // checkCUDA(hipDeviceSynchronize());
  T *host_ptr;
  checkCUDA(hipHostAlloc(&host_ptr,
                          sizeof(T) * num_elements,
                          hipHostMallocPortable | hipHostMallocMapped));
  checkCUDA(hipMemcpy(
      host_ptr, ptr, sizeof(T) * num_elements, hipMemcpyDeviceToHost));
  // checkCUDA(hipDeviceSynchronize());
  int idx = 0;
  printf("%s", prefix);
  for (idx = 0; idx < num_elements; idx++) {
    printf(" %.4lf", (float)host_ptr[idx]);
    if (idx >= 16) {
      break;
    }
  }
  printf("\n");
  checkCUDA(hipHostFree(host_ptr));
}

ffStatus_t
    cudnnSetTensorDescriptorFromArrayShape(hipdnnTensorDescriptor_t tensor,
                                           ArrayShape const &shape) {
  std::vector<std::size_t> reversed_dims(shape.dims.begin(), shape.dims.end());
  reversed(reversed_dims);
  ArrayShape flipped(reversed_dims);

  if (flipped.get_dim() == 5) {
    assert(flipped[legion_dim_t(0)] == 1);
    flipped = flipped.sub_shape(legion_dim_t(1), std::nullopt);
  }

  assert(flipped.get_dim() > 0);
  assert(flipped.get_dim() < 4);

  return hipdnnSetTensor4dDescriptor(tensor,
                                    HIPDNN_TENSOR_NCHW,
                                    HIPDNN_DATA_FLOAT,
                                    flipped.at_maybe(0).value_or(1),
                                    flipped.at_maybe(1).value_or(2),
                                    flipped.at_maybe(2).value_or(3),
                                    flipped.at_maybe(3).value_or(3));
}

hipdnnDataType_t ff_to_cudnn_datatype(DataType type) {
  switch (type) {
    case DataType::FLOAT:
      return HIPDNN_DATA_FLOAT;
    case DataType::DOUBLE:
      return HIPDNN_DATA_DOUBLE;
    case DataType::INT32:
      return HIPDNN_DATA_INT32;
    default:
      assert(false && "Unsupported cudnn data type");
  }
  return HIPDNN_DATA_FLOAT;
}

hipDataType ff_to_cuda_datatype(DataType type) {
  switch (type) {
    case DataType::FLOAT:
      return HIP_R_32F;
    case DataType::DOUBLE:
      return HIP_R_64F;
    case DataType::INT32:
      return HIP_R_32I;
    default:
      assert(false && "Unspoorted cuda data type");
  }
  return HIP_R_32F;
}

template <DataType DT>
struct AssignKernel {
  void operator()(void *ptr, size_t size, void *value) const {
    using ValueType = real_type<DT>;
    ValueType val = *static_cast<ValueType *>(value);
    assign_kernel<ValueType><<<GET_BLOCKS(size), CUDA_NUM_THREADS>>>(
        static_cast<ValueType *>(ptr), size, val);
  }
};

void dispatch_assign_kernel(DataType type,
                            void *ptr,
                            size_t size,
                            void *value) {
  DataTypeDispatch1<AssignKernel>{}(type, ptr, size, value);
}

template <DataType DT>
struct AddKernel {
  void operator()(void *dst, void const *src, size_t size) const {
    using ValueType = real_type<DT>;
    add_kernel<ValueType><<<GET_BLOCKS(size), CUDA_NUM_THREADS>>>(
        static_cast<ValueType *>(dst),
        static_cast<ValueType const *>(src),
        size);
  }
};

void dispatch_add_kernel(DataType type,
                         void *dst,
                         void const *src,
                         size_t size) {
  DataTypeDispatch1<AddKernel>{}(type, dst, src, size);
}

template <DataType DT>
struct CopyKernel {
  void operator()(void *dst, void const *src, coord_t size) const {
    using ValueType = real_type<DT>;
    copy_kernel<ValueType><<<GET_BLOCKS(size), CUDA_NUM_THREADS>>>(
        static_cast<ValueType *>(dst),
        static_cast<ValueType const *>(src),
        size);
  }
};

void dispatch_copy_kernel(DataType type,
                          void *dst,
                          void const *src,
                          coord_t size) {
  DataTypeDispatch1<CopyKernel>{}(type, dst, src, size);
}

template <DataType DT>
struct ApplyAddWithScaleKernel {
  void operator()(void *data_ptr,
                  void const *grad_ptr,
                  size_t size,
                  float scale) const {
    using ValueType = real_type<DT>;
    apply_add_with_scale<ValueType><<<GET_BLOCKS(size), CUDA_NUM_THREADS>>>(
        static_cast<ValueType *>(data_ptr),
        static_cast<ValueType const *>(grad_ptr),
        size,
        scale);
  }
};

void dispatch_apply_add_with_scale_kernel(DataType type,
                                          void *data_ptr,
                                          void const *grad_ptr,
                                          size_t size,
                                          float scale) {
  DataTypeDispatch1<ApplyAddWithScaleKernel>{}(
      type, data_ptr, grad_ptr, size, scale);
}

template __host__ void
    print_tensor<float>(float const *ptr, size_t rect, char const *prefix);
template __host__ void
    print_tensor<double>(double const *ptr, size_t rect, char const *prefix);
template __host__ void
    print_tensor<int32_t>(int32_t const *ptr, size_t rect, char const *prefix);
template __host__ void
    print_tensor<int64_t>(int64_t const *ptr, size_t rect, char const *prefix);
